#include "cudamat.cuh"
#include "cudaconv2/conv_util.cuh"
#include "cudaconv2/cudaconv2.cuh"
#include "nvmatrix/nvmatrix.cuh"

/*
 * images:      (numImgColors, imgPixels, numImages)
 * filters:     (numFilterColors, filterPixels, numFilters)
 * targets:     (numFilters, numModules, numImages)
 */

// Convolutions.
extern "C" void convUp(cudamat* images, cudamat* filters, cudamat* targets, int numModulesX, int paddingStart, int moduleStride, int numImgColors, int numGroups){
  _filterActsCu(images, filters, targets, numModulesX, paddingStart, moduleStride, numImgColors, numGroups, 0, 1, true);
}
extern "C" void convDown(cudamat* images, cudamat* filters, cudamat* targets, int imgSize, int paddingStart, int moduleStride, int numImgColors, int numGroups){
  _imgActsCu(images, filters, targets, imgSize, paddingStart, moduleStride, numImgColors, numGroups, 0, 1, true);
}
extern "C" void convOutp(cudamat* images, cudamat* hidSums, cudamat* targets, int numModulesX, int filterSize, int paddingStart, int moduleStride, int numImgColors, int numGroups, int partialSum){
  _weightActsCu(images, hidSums, targets, numModulesX, filterSize, paddingStart, moduleStride, numImgColors, numGroups, partialSum, 0, 1);
}

// Local Connections.
extern "C" void localUp(cudamat* images, cudamat* filters, cudamat* targets, int numModulesX, int paddingStart, int moduleStride, int numImgColors, int numGroups){
  _filterActsCu(images, filters, targets, numModulesX, paddingStart, moduleStride, numImgColors, numGroups, 0, 1, false);
}
extern "C" void localDown(cudamat* images, cudamat* filters, cudamat* targets, int imgSize, int paddingStart, int moduleStride, int numImgColors, int numGroups){
  _imgActsCu(images, filters, targets, imgSize, paddingStart, moduleStride, numImgColors, numGroups, 0, 1, false);
}
extern "C" void localOutp(cudamat* images, cudamat* hidSums, cudamat* targets, int numModulesX, int filterSize, int paddingStart, int moduleStride, int numImgColors, int numGroups, int partialSum){
  _weightActsCu(images, hidSums, targets, numModulesX, filterSize, paddingStart, moduleStride, numImgColors, numGroups, 1, 0, 1);
}

// Response Normalization.
extern "C" void ResponseNorm(cudamat* images, cudamat* denoms, cudamat* targets, int numFilters, int sizeX, float addScale, float powScale){
  convResponseNormCu(images, denoms, targets, numFilters, sizeX, addScale,  powScale);
}

extern "C" void ResponseNormUndo(cudamat* outGrads, cudamat* denoms, cudamat* inputs, cudamat* acts, cudamat* targets, int numFilters, int sizeX, float addScale, float powScale){
  convResponseNormUndoCu(outGrads, denoms, inputs, acts, targets, numFilters, sizeX, addScale, powScale, 0, 1);
}

// Contrast Normalization.
extern "C" void ContrastNorm(cudamat* images, cudamat* meanDiffs, cudamat* denoms, cudamat* targets, int numFilters, int sizeX, float addScale, float powScale){
  convContrastNormCu(images, meanDiffs, denoms, targets, numFilters, sizeX, addScale,  powScale);
}

extern "C" void ContrastNormUndo(cudamat* outGrads, cudamat* denoms, cudamat* meanDiffs, cudamat* acts, cudamat* targets, int numFilters, int sizeX, float addScale, float powScale){
  convContrastNormUndoCu(outGrads, denoms, meanDiffs, acts, targets, numFilters, sizeX, addScale, powScale, 0, 1);
}

// Pooling.
extern "C" void MaxPool(cudamat* images, cudamat* targets, int numFilters, int subsX,	int startX,	int strideX, int outputsX){
  MaxPooler mpooler;
  convLocalPoolCu<MaxPooler>(images, targets, numFilters, subsX, startX, strideX, outputsX, mpooler);
}
extern "C" void ProbMaxPool(cudamat* images, cudamat* rnd, cudamat* targets, int numFilters, int subsX,	int startX,	int strideX, int outputsX){
  ProbMaxPooler mpooler;
  convLocalProbPoolCu<ProbMaxPooler>(images, rnd, targets, numFilters, subsX, startX, strideX, outputsX, mpooler);
}


extern "C" void MaxPoolUndo(cudamat* images, cudamat* maxGrads, cudamat* maxActs, cudamat* targets, int subsX, int startX, int strideX, int outputsX){
  convLocalMaxUndoCu(images, maxGrads, maxActs, targets, subsX, startX, strideX, outputsX, 0, 1);
}

