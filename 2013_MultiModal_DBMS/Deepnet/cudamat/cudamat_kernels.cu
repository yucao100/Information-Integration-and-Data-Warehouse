#include "hip/hip_runtime.h"
#include "cudamat_kernels.cuh"
#include "float.h"
/* ------------------------- Random number generation ------------------------- */

__global__ void kSeedRandom(unsigned int* rndMults, unsigned long long* rndWords, unsigned int seed) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // The initial x is the seed and the initial carry is 1
    unsigned long long rndWord = ((unsigned long long)seed << 32) + 1;
    const unsigned int rndMult = rndMults[idx];
    /*
     * Run the chain for a few steps so that all the streams have a chance
     * to differentiate. They start out generating similar random numbers
     * because all the multipliers are similar.
     */
    for(unsigned int i = 0; i < NUM_RND_BURNIN; i++) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    }
    rndWords[idx] = rndWord;
}

__global__ void kRandomUniform(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];

    for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        gData[i] = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    }
    rndWords[idx] = rndWord;
}

__global__ void kRandomGaussian(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];

    float rnd1, rnd2, R, T;
    for(unsigned int i = idx; i < numElements; i += 2*NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        rnd1 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        rnd2 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
        T = 2 * PI * rnd2;
        R = sqrtf(-2 * __logf(rnd1));
        gData[i] = R * __cosf(T);
        if (i + NUM_RND_STREAMS < numElements)
            gData[i + NUM_RND_STREAMS] = R * __sinf(T);
    }
    rndWords[idx] = rndWord;
}

__global__ void kRandomDropout(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements, float dropprob, float val) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];

    for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        gData[i] = ((__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f) > dropprob ? gData[i]:val;
    }
    rndWords[idx] = rndWord;
}

__global__ void kSampleBernoulli(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];

    for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        target[i] = ((__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f) < gData[i] ? 1:0;
    }
    rndWords[idx] = rndWord;
}
__global__ void kSampleBernoulliTanh(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];

    for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        target[i] = ((__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f) < (1.0 + gData[i]) / 2.0 ? 1:0;
    }
    rndWords[idx] = rndWord;
}

__global__ void kSamplePoisson(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];

    for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        target[i] = gData[i];
    }
    rndWords[idx] = rndWord;
}

__global__ void kSampleGaussian(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements, float mult) {

    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];

    float rnd1, rnd2, R, T;
    for(unsigned int i = idx; i < numElements; i += 2*NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        rnd1 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        rnd2 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
        T = 2 * PI * rnd2;
        R = sqrtf(-2 * __logf(rnd1));
        target[i] = gData[i] + mult * R * __cosf(T);
        if (i + NUM_RND_STREAMS < numElements)
            target[i + NUM_RND_STREAMS] = gData[i + NUM_RND_STREAMS] + mult * R * __sinf(T);
    }
    rndWords[idx] = rndWord;
}

__global__ void kPerturbEnergy(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];
    float rnd;

    for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        rnd = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
        target[i] = gData[i] - __logf( - __logf(rnd));
    }
    rndWords[idx] = rndWord;
}

__global__ void kPerturbProb(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];
    float rnd;

    for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        rnd = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
        target[i] = - gData[i] /  __logf(rnd);
    }
    rndWords[idx] = rndWord;
}


/* ------------------------- Data copying ------------------------- */

/*
Copy row slice from source to target. There is a block for every 32x32 chunk being copied.
*/
__global__ void kGetRowSlice(float* source, float* target, int start, int end, int width, int height) {
    const int row = start + blockIdx.x * 32 + threadIdx.x;
    const int start_col = blockIdx.y * 32;

    const int end_col = (start_col + 32 < width) ? start_col + 32: width;

    const int target_height = end - start;

    if (row < end) {
        for (int cur_col = start_col; cur_col < end_col; cur_col++)
            target[cur_col * target_height + row - start] = source[cur_col * height + row];
    }
}

__global__ void kSetRowSlice(float* source, float* target, int start, int end, int width, int height) {
    const int row = start + blockIdx.x * 32 + threadIdx.x;
    const int start_col = blockIdx.y * 32;

    const int end_col = (start_col + 32 < width) ? start_col + 32: width;

    const int source_height = end - start;

    if (row < end) {
        for (int cur_col = start_col; cur_col < end_col; cur_col++)
            target[cur_col * height + row] = source[cur_col * source_height + row - start];
            //source[cur_col * height + row - start] = target[cur_col * target_height + row];
    }
}

__global__ void kTranspose(float *odata, float *idata, int width, int height) {
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < width) && (yIndex < height)) {
        unsigned int index_in = yIndex * width + xIndex;

        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < height) && (yIndex < width)) {
        unsigned int index_out = yIndex * height + xIndex;

        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}

/* ------------------------- Mathematical operations ------------------------- */

__global__ void kLessThan(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat1[i] < mat2[i];
    }
}

__global__ void kLessThanScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] < val;
    }
}

__global__ void kGreaterThan(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat1[i] > mat2[i];
    }
}

__global__ void kUpperBound(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat1[i] > mat2[i] ? mat2[i] : mat1[i];
    }
}

__global__ void kLowerBound(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat1[i] < mat2[i] ? mat2[i] : mat1[i];
    }
}


__global__ void kGreaterThanScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] > val;
    }
}

__global__ void kUpperBoundScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] > val ? val:mat[i];
    }
}
__global__ void kLowerBoundScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] < val ? val:mat[i];
    }
}

__global__ void kSumAll(float* mat, unsigned int len) {
    __shared__ float sum_vals[32];
    float val = 0;
 
    for (unsigned int i = threadIdx.x; i < len; i += 32)
        val += mat[i];

    sum_vals[threadIdx.x] = val;

    __syncthreads();

    if (threadIdx.x == 0) {
        val = 0;
        for (unsigned int i = 0; i < 32; i++)
            val += sum_vals[i];
        device_val = val;
    }
}

__global__ void kCumsum(float *mat, float *target, float *temp, unsigned int height) { 
  // extern  __shared__  float temp[];// allocated on invocation
  const int thid = threadIdx.x;
  if (2*thid < height) {
    const int super_offset = blockIdx.x * height;
    target += super_offset;
    mat += super_offset;
    temp += super_offset;
    int offset = 1;
    //float s = 0.0;
    temp[2*thid]   = mat[2*thid]; // load input into shared memory 
    temp[2*thid+1] = mat[2*thid+1]; 
    for (int d = height>>1; d > 0; d >>= 1) {// build sum in place up the tree 
      __syncthreads(); 
      if (thid < d) {
        int ai = offset*(2*thid+1)-1; 
        int bi = offset*(2*thid+2)-1; 
        temp[bi] += temp[ai];
      } else if (thid == d && thid % 2 == 1) {
        //int bi = offset*(2*thid+2)-1; 
        //temp[bi] += temp[ai];

      }

      offset *= 2;
    }
    if (thid == 0) { temp[height - 1] = 0; } // clear the last element 
    for (int d = 1; d < height; d *= 2)  { // traverse down tree & build scan 
      offset >>= 1; 
      __syncthreads(); 
      if (thid < d) {
        int ai = offset*(2*thid+1)-1; 
        int bi = offset*(2*thid+2)-1; 
        float t   = temp[ai]; 
        temp[ai]  = temp[bi]; 
        temp[bi] += t; 
      } 
    } 
    __syncthreads();
    // write results to device memory 
  //  if (thid == -1) {
  //    target[0]   = temp[1];
  //    target[height-1] = s;
  //  } else {
      target[2*thid]   = temp[2*thid];
      target[2*thid+1] = temp[2*thid+1];
  //  }
  }

}


__global__ void kMaxColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
    __shared__ float max_vals[32];
    float cur_max = -FLT_MAX;
    float val = 0;
 
    for (unsigned int i = threadIdx.x; i < height; i += 32) {
        val = mat[blockIdx.x * height + i];

        if (val > cur_max)
            cur_max = val;
    }

    max_vals[threadIdx.x] = cur_max;

    __syncthreads();

    if (threadIdx.x == 0) {
        cur_max = -FLT_MAX;

        for (unsigned int i = 0; i < 32; i++)
            if (max_vals[i] > cur_max)
                cur_max = max_vals[i];

        target[blockIdx.x] = cur_max;
    }
}

__global__ void kChooseMaxAndAccumulate(float* mat, float* acc, unsigned int height) {
    __shared__ float max_vals[32];
    __shared__ unsigned int max_val_args[32];
    float cur_max = -FLT_MAX;
    unsigned int cur_argmax = 0;
    float val = 0;
 
    for (unsigned int i = threadIdx.x; i < height; i += 32) {
        val = mat[blockIdx.x * height + i];
        if (val > cur_max) {
            cur_max = val;
            cur_argmax = i;
        }
    }

    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;

    __syncthreads();

    if (threadIdx.x == 0) {
        cur_max = -FLT_MAX;
        cur_argmax = 0;

        for (unsigned int i = 0; i < 32; i++)
            if (max_vals[i] > cur_max) {
                cur_max = max_vals[i];
                cur_argmax = max_val_args[i];
            }
        acc[blockIdx.x * height + cur_argmax] += 1;
    }
}


__global__ void kChooseMaxColumnwise(float* mat, float* target, unsigned int height) {
    __shared__ float max_vals[32];
    __shared__ unsigned int max_val_args[32];
    float cur_max = -FLT_MAX;
    unsigned int cur_argmax = 0;
    float val = 0;
 
    for (unsigned int i = threadIdx.x; i < height; i += 32) {
        val = mat[blockIdx.x * height + i];
        target[blockIdx.x * height + i] = 0;

        if (val > cur_max) {
            cur_max = val;
            cur_argmax = i;
        }
    }

    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;

    __syncthreads();

    if (threadIdx.x == 0) {
        cur_max = -FLT_MAX;
        cur_argmax = 0;

        for (unsigned int i = 0; i < 32; i++)
            if (max_vals[i] > cur_max) {
                cur_max = max_vals[i];
                cur_argmax = max_val_args[i];
            }
        target[blockIdx.x * height + cur_argmax] = 1;
    }
}


__global__ void kArgMaxColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
    __shared__ float max_vals[32];
    __shared__ unsigned int max_val_args[32];
    float cur_max = -FLT_MAX;
    float cur_argmax = 0;
    float val = 0;
 
    for (unsigned int i = threadIdx.x; i < height; i += 32) {
        val = mat[blockIdx.x * height + i];

        if (val > cur_max) {
            cur_max = val;
            cur_argmax = i;
        }
    }

    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;

    __syncthreads();

    if (threadIdx.x == 0) {
        cur_max = -FLT_MAX;
        cur_argmax = 0;

        for (unsigned int i = 0; i < 32; i++)
            if (max_vals[i] > cur_max) {
                cur_max = max_vals[i];
                cur_argmax = max_val_args[i];
            }   
        target[blockIdx.x] = cur_argmax;
    }
}

__global__ void kSqSumColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
    __shared__ float sum_vals[32];
    float cur_sum = 0;
 
    for (unsigned int i = threadIdx.x; i < height; i += 32) {
        cur_sum += mat[blockIdx.x * height + i] * mat[blockIdx.x * height + i];
    }

    sum_vals[threadIdx.x] = cur_sum;

    __syncthreads();

    if (threadIdx.x == 0) {
        cur_sum = 0;

        for (unsigned int i = 0; i < 32; i++)
            cur_sum += sum_vals[i];

        target[blockIdx.x] = cur_sum;
    }
}

__global__ void kNormLimitColumnwise(float* mat, float* target,
                                     float norm, unsigned int width,
                                     unsigned int height) {
    __shared__ float sum_vals[33];
    float cur_sum = 0;
    for (unsigned int i = threadIdx.x; i < height; i += 32) {
        cur_sum += mat[blockIdx.x * height + i] * mat[blockIdx.x * height + i];
    }
    sum_vals[threadIdx.x] = cur_sum;
    __syncthreads();
    if (threadIdx.x == 0) {
        cur_sum = 0;
        for (unsigned int i = 0; i < 32; i++)
            cur_sum += sum_vals[i];
        sum_vals[32] = norm > cur_sum ? 1 : norm / sqrt(cur_sum);
    }
    float scale = sum_vals[32];
    for (unsigned int i = threadIdx.x; i < height; i += 32) {
        target[blockIdx.x * height + i] = mat[blockIdx.x * height + i] * scale;
    }
    __syncthreads();
}


__global__ void kSign(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] ? copysignf(1., mat[i]) : 0.;
    }
}

__global__ void kApplySin(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = __sinf(mat[i]);
    }
}

__global__ void kApplyCos(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = __cosf(mat[i]);
    }
}

__global__ void kApplySigmoid(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = 1 / (1 + __expf(-mat[i]));
    }
}


__global__ void kApplyTanh(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    float mat_i, exp2x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        mat_i = mat[i];
        exp2x = __expf(2 * mat_i);
        target[i] = 1 - 2 / (exp2x + 1);
    }
}

__global__ void kApplyAbs(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] * ((mat[i] > 0) - (mat[i] < 0));
    }
}

__global__ void kApplyLog1PlusExpExact(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    float mat_i;

    for (unsigned int i = idx; i < len; i += numThreads) {
        mat_i = mat[i];
        if (mat_i > 0)
            target[i] = (log(1 + exp(-mat_i)) + mat_i);
        else
            target[i] = log(1 + exp(mat_i));
    }
}


__global__ void kApplyLog1PlusExp(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    float mat_i;

    for (unsigned int i = idx; i < len; i += numThreads) {
        mat_i = mat[i];
        if (mat_i > 0)
            target[i] = (__logf(1 + __expf(-mat_i)) + mat_i);
        else
            target[i] = __logf(1 + __expf(mat_i));
    }
}

__global__ void kLog(float* mat, float* target, unsigned int len, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = __logf(mat[i] + tiny);
    }
}

__global__ void kExp(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = __expf(mat[i]);
    }
}
__global__ void kCeil(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = ceil(mat[i]);
    }
}
__global__ void kFloor(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = floor(mat[i]);
    }
}



__global__ void kSqrt(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = sqrt(mat[i]);
    }
}

__global__ void kPow(float* mat, float pow, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = powf(mat[i], pow);
    }
}

__global__ void kPowMatrix(float* mat, float* pow, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = powf(mat[i], pow[i]);
    }
}

__global__ void kCrossEntropy(float* mat, float* p, float* target, unsigned int len, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = -mat[i] * __logf(p[i] + tiny);
    }
}
__global__ void kCrossEntropyBernoulli(float* mat, float* p, float* target, unsigned int len, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = -mat[i] * __logf(p[i] + tiny) - (1 - mat[i]) * __logf(1 - p[i] + tiny);
    }
}

__global__ void kCorrectPreds(float* mat, float* p, float* target, unsigned int len, float cutoff) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] * (p[i] >= cutoff) + (1 - mat[i]) * (p[i] < cutoff);
    }
}

__global__ void kReciprocal(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads)
        target[i] = 1. / mat[i];
}

__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, unsigned int width,
                              unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] + vec[i % height];
    }
}
__global__ void kAddDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width; i += numThreads) {
        tgtMat[width*i + i] = mat[width*i + i] + val;
    }
}

__global__ void kAddDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width; i += numThreads) {
        tgtMat[width*i + i] = mat[width*i + i] + vec[i];
    }
}

__global__ void kMultDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width; i += numThreads) {
        tgtMat[width*i + i] = mat[width*i + i] * val;
    }
}

__global__ void kMultDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width; i += numThreads) {
        tgtMat[width*i + i] = mat[width*i + i] * vec[i];
    }
}
__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] + vec[i / height];
    }
}

__global__ void kAddColMult(float* mat, float* vec, float* tgtMat, float mult,
                            unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] + mult * vec[i % height];
    }
}

__global__ void kAddRowMult(float* mat, float* vec, float* tgtMat, float mult, 
                            unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] + mult * vec[i / height];
    }
}
__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] * vec[i % height];
    }
}

__global__ void kDivByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] / vec[i / height];
    }
}

__global__ void kDivByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] / vec[i % height];
    }
}

__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] * vec[i / height];
    }
}
__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] + b[i];
    }
}

__global__ void kSubtract(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] - b[i];
    }
}

__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] / b[i];
    }
}

__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] * b[i];
    }
}

__global__ void kCosDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = -a[i] * __sinf(b[i]);
    }
}

__global__ void kSinDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] * __cosf(b[i]);
    }
}

__global__ void kLogisticDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] * b[i] * (1.0 - b[i]);
    }
}

__global__ void kTanhDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] * (1.0 + b[i]) * (1.0 - b[i]) * 0.5;
    }
}

__global__ void kRectifiedLinearDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] * (b[i] > 0 ? 1 : 0);
    }
}

__global__ void kRectifiedLinearSmoothDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] * (1 - __expf(-b[i]));
    }
}

__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        dest[i] = alpha * mat[i];
    }
}

__global__ void kAssignScalar(float* dest, float alpha, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        dest[i] = alpha;
    }
}

__global__ void kDivideScalar(float* mat, float alpha, float* dest, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        dest[i] = mat[i] / alpha;
    }
}

__global__ void kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] + alpha;
    }
}

__global__ void kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows){
    __shared__ int sourceRowIndices[32];
    const int startTargetRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs-startTargetRowI);

    // cooperatively load 32 row indices
    if (tid < localNRowIs){
        sourceRowIndices[tid] = int(indices[startTargetRowI + tid]);
        if (sourceRowIndices[tid]<0)
            sourceRowIndices[tid] += nSourceRows;
        if (sourceRowIndices[tid]<0 || sourceRowIndices[tid]>=nSourceRows)
            sourceRowIndices[tid] = -1;
    }
    __syncthreads();

    // copy 32 rows
    for (int i=0; i<localNRowIs; i++){
        const int targetRowI = startTargetRowI + i, sourceRowI = sourceRowIndices[i];
        for (int colI=tid; colI<nCols; colI+=32)
            target[targetRowI * nCols + colI] = sourceRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
    }
}

__global__ void kSwapRows(float* source, float* target, float* indices1, float* indices2, int nRowIs, int nCols, int nRows){
    __shared__ int sourceRowIndices[32], targetRowIndices[32];
    const int startRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs-startRowI);

    // cooperatively load 32 row indices
    if (tid < localNRowIs){
        sourceRowIndices[tid] = int(indices1[startRowI + tid]);
        targetRowIndices[tid] = int(indices2[startRowI + tid]);
        if (sourceRowIndices[tid]<0)
            sourceRowIndices[tid] += nRows;
        if (sourceRowIndices[tid]<0 || sourceRowIndices[tid]>=nRows)
            sourceRowIndices[tid] = -1;
        if (targetRowIndices[tid]<0)
            targetRowIndices[tid] += nRows;
        if (targetRowIndices[tid]<0 || targetRowIndices[tid]>=nRows)
            targetRowIndices[tid] = -1;
    }
    __syncthreads();

    // copy 32 rows
    for (int i=0; i<localNRowIs; i++){
        const int sourceRowI = sourceRowIndices[i], targetRowI = targetRowIndices[i];
        for (int colI=tid; colI<nCols; colI+=32) {
            const float temp1 = sourceRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
            const float temp2 = targetRowI==-1 ? (1.0/0.0 -1.0/0.0) : target[targetRowI * nCols + colI];
            if (sourceRowI != -1)
              source[sourceRowI * nCols + colI] =  temp2;
            if (targetRowI != -1)
              target[targetRowI * nCols + colI] = temp1;
        }
    }
}

__global__ void kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows){
    __shared__ int targetRowIndices[32];
    const int startSourceRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs-startSourceRowI);

    // cooperatively load 32 row indices
    if (tid < localNRowIs){
        targetRowIndices[tid] = int(indices[startSourceRowI + tid]);
        if (targetRowIndices[tid]<0)
            targetRowIndices[tid] += nTargetRows;
        if (targetRowIndices[tid]<0 || targetRowIndices[tid]>=nTargetRows)
            targetRowIndices[tid] = -1;
    }
    __syncthreads();

    // copy 32 rows
    for (int i=0; i<localNRowIs; i++){
        const int sourceRowI = startSourceRowI + i, targetRowI = targetRowIndices[i];
        for (int colI=tid; colI<nCols; colI+=32)
            target[targetRowI * nCols + colI] = targetRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
    }
}

__global__ void kBlockify(float* source, float* target, int numdims, int blocksize) {
    const unsigned int idx = threadIdx.x;
    const unsigned int numThreads = blockDim.x;
    const int off = blockIdx.x * numdims;

    for (unsigned int target_ind = idx; target_ind < numdims; target_ind += numThreads) {
        const int block = target_ind / blocksize;
        target[off + target_ind] = source[off + block * blocksize];
    }
}

__global__ void kGenerateTranslationsBigVarOff(float* source, float* target, float* off_x_arr, float* off_y_arr, int source_w, int target_w, int num_channels) {
    const unsigned int idx = threadIdx.x;
    const unsigned int numThreads = blockDim.x;

    int target_x, target_y;
    int pad = (source_w - target_w)/2;
    int target_tile_size = target_w * target_w;
    int source_tile_size = source_w * source_w;

    int off_x = off_x_arr[blockIdx.x];
    int off_y = off_y_arr[blockIdx.x];
    int target_off = blockIdx.x * target_tile_size;
    int source_off = blockIdx.x * source_tile_size + (pad + off_x) * source_w + (pad + off_y);

    for (unsigned int target_ind = idx; target_ind < target_tile_size; target_ind += numThreads) {
        target_x = target_ind / target_w;
        target_y = target_ind - target_x * target_w;

        for (unsigned int ch = 0; ch < num_channels; ch += 1) {
            target[num_channels*(target_off + target_x * target_w + target_y) + ch] = source[num_channels*(source_off + target_x * source_w + target_y) + ch];
        }
    }
}

